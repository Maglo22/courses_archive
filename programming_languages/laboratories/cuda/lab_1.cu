// compile -> nvcc lab_1.cu -o lab_1
// execute -> lab_1.exe | lab_1.out

// Bruno Maglioni A01700879


#include <hip/hip_runtime.h>
#include <iostream>

#define N 100000 // Size of problem
#define TPB 512 // Threads per Block


// Calculates PI sequentially (Riemann Sum) using the CPU.
double cpuPI(long num_rects, double width){
  long i;
  double mid, height, area;
  double sum = 0.0;

  for (i = 0; i < num_rects; i++) {
    mid = (i + 0.5) * width;
    height = 4.0 / (1.0 + mid * mid);
    sum += height;
  }

  area = sum * width;

  return area;
}

// Adds values in array
double addArr(double *arr, double width){
  long i;
  double sum = 0.0;

  for (i = 0; i < N; i++) {
    sum += arr[i];
  }

  return sum;
}

// Calculates PI parallely (Riemann Sum) using the GPU.
__global__ void gpuPI(double *res, double width, long max){
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int id = index;
  double mid;

  while(id < max){
    mid = (id + 0.5) * width;
    res[id] = (4.0 / (1.0 + mid * mid)) * width;

    id = id + blockDim.x * gridDim.x;
  }
}

int main(){
  double *res; // CPU variables
  double *d_res; // GPU variables

  double piCPU, piGPU, width = 1.0 / N;

  // Allocate memory on CPU
  res = (double*) malloc(sizeof(double) * N); // Result Array

  // Allocate memory on GPU
  hipMalloc((void**)&d_res, sizeof(double) * N);

  // Call function in GPU
  gpuPI<<< (N / TPB), TPB>>>(d_res, width, N);

  // Copy result array from GPU to CPU
  hipMemcpy(res, d_res, N * sizeof(double), hipMemcpyDeviceToHost);

  piCPU = cpuPI(N, width); // Calculate PI using the CPU
  piGPU = addArr(res, width); // Calculate PI by adding the array returned by GPU

  printf("Pi using CPU: %f\n", piCPU);
  printf("Pi using GPU: %f\n", piGPU);

  // Free CPU memory
  free(res);

  // Free GPU memory
  hipFree(d_res);

  return 0;
}
