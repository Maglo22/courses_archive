// compile -> nvcc lab_2.cu -o lab_2
// execute -> lab_2.exe | lab_2.out

// Bruno Maglioni A01700879


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define N 4
#define ThreadsPerBlock N * N
#define NumBlocks  ceil((ThreadsPerBlock + (N * N)) / ThreadsPerBlock)


__global__ void multi(double *a, double *b, double *c, int n){
  int row = threadIdx.y + blockIdx.y * blockDim.y;
  int col = threadIdx.x + blockIdx.x * blockDim.x;

  double sum = 0.0;

  if(row < n && col < n){
    for(int i = 0; i < n; i++){
      sum += a[row * n + i] * b[i * n + col];
    }
  }
  c[row * n + col] = sum;
}

// matrix multiplication using CPU
// a = M x N, b = P x Q
// m = rows of first matrix, p = rows of second matrix, q = columns of second matrix
void mat_multi(double *a, double *b, double *c, int m, int p, int q){
  int i, j, k;
  double sum = 0.0;

  for (i = 0; i < m; i++) {
    for (j = 0; j < q; j++) {
      for (k = 0; k < p; k++) {
        sum += a[i * m + k] * b[k * m + j];
      }

      c[i * m + j] = sum;
      sum = 0.0;
    }
  }
}


// Fills matrix
void fill_mat(double* mat){
  for(int i = 0; i < N; i++){
    for(int j = 0; j < N; j++){
      mat[i * N + j] = rand() % 50;
    }
  }
}


// Prints matrix
void print_mat(double* mat){
  int i, j;
  for(i = 0; i < N; i++){
    for(j = 0; j < N; j++){
      printf("%f\t", mat[i * N + j]);
    }
    printf("\n");
  }
  printf("\n");
}


int main(){
  double *mat_1, *mat_2, *res; // CPU variables
  double *d_mat_1, *d_mat_2, *d_res; // GPU variables

  // for random number generation
  time_t t;
  srand((unsigned) time(&t));

  // Allocate memory on CPU
  mat_1 = (double*) malloc(sizeof(double) * N * N); // Matrix 1
  mat_2 = (double*) malloc(sizeof(double) * N * N); // Matrix 2
  res = (double*) malloc(sizeof(double) * N * N); // Result Matrix

  // Allocate memory on GPU
  hipMalloc((void**)&d_mat_1, sizeof(double) * N * N);
  hipMalloc((void**)&d_mat_2, sizeof(double) * N * N);
  hipMalloc((void**)&d_res, sizeof(double) * N * N);

  fill_mat(mat_1);
  fill_mat(mat_2);

  printf("Matrix 1:\n");
  print_mat(mat_1);

  printf("\nMatrix 2:\n");
  print_mat(mat_2);

  // Copy CPU variables to GPU
  hipMemcpy(d_mat_1, mat_1, sizeof(double) * N * N, hipMemcpyHostToDevice);
  hipMemcpy(d_mat_2, mat_2, sizeof(double) * N * N, hipMemcpyHostToDevice);

  // create 2D grid
  dim3 blocks(NumBlocks, NumBlocks); // dimensions of resulting matrix
  dim3 threads(ThreadsPerBlock, ThreadsPerBlock);

  // Call function in GPU
  multi<<<blocks, threads>>>(d_mat_1, d_mat_2, d_res, N);

  // Copy result matrix from GPU to CPU
  hipMemcpy(res, d_res, sizeof(double) * N * N, hipMemcpyDeviceToHost);

  printf("\nResult matrix:\n");
  print_mat(res);

  // Free CPU memory
  free(mat_1);
  free(mat_2);
  free(res);

  // Free GPU memory
  hipFree(d_mat_1);
  hipFree(d_mat_2);
  hipFree(d_res);

  return 0;
}
