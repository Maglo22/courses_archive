// compile -> nvcc quiz3.cu -o quiz3
// execute -> quiz3.exe | quiz3.out

// Bruno Maglioni A01700879


#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 9		//size of original matrix
#define K N/3		//size of compressed matrrix
#define ThreadsPerBlock N/K
#define NumBlocks N/K

__global__ void compress(float *mat, int n, float *comp, int k){
  int x = threadIdx.x + blockIdx.x * blockDim.x; // columns
  int y = threadIdx.y + blockIdx.y * blockDim.y; // rows
  int offset = x + y * blockDim.x * gridDim.x; // where the thread is on the grid

  if(x < K && y < K){
    for(int i = 0; i < y; i++){
      for(int j = 0; j < x; j++){
        comp[j + (i * k)] += mat[offset + (j + (i * n))]/n;
      }
    }
    offset += blockDim.x * gridDim.x;
  }
}

void print_mat(float *mat, int n){
	for (int i = 0; i < n; i++){
		for (int j = 0; j < n; j++){
			printf("%.1f\t", mat[i*n+j]);
		}
		printf("\n");
	}
	printf("\n");
}


void fill_mat(float *mat, int n){
	int c = 0;
	for (int i = 0; i < n; i++){
		for (int j = 0; j < n; j++){
			mat[i*n+j] = c++;
		}
	}
}

int main(){
	float *h_compress, *h_matrix; // CPU variables
	float *d_compress, *d_matrix; // GPU variables

  // Allocate memory on CPU
	h_compress = (float *)malloc(sizeof(float) * K * K);
	h_matrix = (float *)malloc(sizeof(float) * N * N);

  // Allocate memory on GPU
  hipMalloc((void**)&d_compress, sizeof(float) * K * K);
  hipMalloc((void**)&d_matrix, sizeof(float) * N * N);

  // Fill matrix
  fill_mat(h_matrix, N);

	printf("\n input mat \n");
	print_mat(h_matrix, N);

  // Copy CPU variables to GPU
  hipMemcpy(d_compress, h_compress, sizeof(float)* K * K, hipMemcpyHostToDevice);
  hipMemcpy(d_matrix, h_matrix, sizeof(float)* N * N, hipMemcpyHostToDevice);

  // Create grids
  dim3 blocks(NumBlocks, NumBlocks);
  dim3 threads(ThreadsPerBlock, ThreadsPerBlock);

  // Call function in GPU
  compress<<<blocks, threads>>>(d_matrix, N, d_compress, K);

  // Copy result matrix from GPU to CPU
  hipMemcpy(h_compress, d_compress, sizeof(float) * K * K, hipMemcpyDeviceToHost);

  // Print compressed matrix
  printf("\n compress mat \n");
	print_mat(h_compress, N);

  // Free CPU memory
  free(h_compress);
  free(h_matrix);

  // Free GPU memory
  hipFree(d_compress);
  hipFree(d_matrix);

  return 0;
}
