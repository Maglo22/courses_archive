#include "hip/hip_runtime.h"
#include <iostream>

const int N = 32 * 1024;
const int Threads_per_block = 256;
const int Blocks_per_grid = imin(32, (N + Threads_per_block - 1) / Threads_per_block); // sets to the smallest between 32 and the operation

__global__ void dot(float* a, float* b, float* c) {
  __shared__ float cache[Threads_per_block]; // cache shared per block (each block has one)

  int id = threadIdx.x + blockIdx.x * blockDim.x; // thread id, moves between blocks
  int cacheId = threadIdx.x; // cache id, represents one block of the grid
  float temp = 0;

  while(id < Blocks_per_grid) {
    temp += a[id] * b[id];
    id += blockDim.x * gridDim.x;
  }

  cache[cacheId] = temp; // set cache values
  __syncthreads(); // synchronize threads in the block

  int i = blockDim.x / 2; // half the block size

  while(i != 0) {
    // each thread adds two of the values in cache, and stores the result back to cache
    if(cacheId < i) {
      cache[cacheId] + = cache[cacheId + i];
    }
    __syncthreads();
    i /= 2;
  }

  if (cacheId == 0) { // the result of every sum of a block is in the first entry of the cache
    c[blockIdx.x] = cache[0];
  }
}

int main(void) {
  float *a, *b, *partial_c, c; // CPU variables
  float *d_a, *d_b, *d_c; // GPU variables

  // Allocate memory on CPU
  a = (float *) malloc(sizeof(float) * N);
  b = (float *) malloc(sizeof(float) * N);
  c = (float *) malloc(sizeof(float) * Blocks_per_grid);

  c = 0;

  // Allocate memory on GPU
  hipMalloc((void**)&d_a, sizeof(float) * N);
  hipMalloc((void**)&d_b, sizeof(float) * N);
  hipMalloc((void**)&d_c, sizeof(float) * Blocks_per_grid);

  // Fill arrays
  for( int i = 0; i < N; i++){
    a[i] = 1;
    b[i] = i * 2;
  }

  // Copy CPU variables to GPU
  hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

  dot<<<Blocks_per_grid, Threads_per_block>>> (d_a, d_b, d_c);

  // Copy result matrix from GPU to CPU
  hipMemcpy(partial_c, d_c, sizeof(float) * N, hipMemcpyDeviceToHost);

  // Add sum values of all the blocks
  for(int i = 0; i < Blocks_per_grid; i++){
    c += partial_c[i];
  }

  printf("Value calculated: %.6g.\n", c);

  // free CPU memory
  free(a);
  free(b);
  free(partial_c);

  // free GPU memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;

}
