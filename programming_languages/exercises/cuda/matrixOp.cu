
#include <hip/hip_runtime.h>
#include <iostream>

#define N 4096
#define TPB 512 // Threads per Block

__global__ void add(int* a, int* b, int *c, int max){
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int id = index;
  while(id < max){
    c[id] = a[id] + b[id];
    id = id + blockDim.x * gridDim.x;
  }
}

// Fills a matrix with 1s
void fill_mat(int* mat){
  int i, j;
  for(i = 0; i < N; i++){
    for(j = 0; j < N; j++){
      mat[i * N + j] = 1;
    }
  }
}

// Prints matrix
void print_mat(int* mat){
  int i, j;
  for(i = 0; i < N; i++){
    for(j = 0; j < N; j++){
      printf("%i\t", mat[i * N + j]);
    }
    printf("\n");
  }
  printf("\n");
}

int main(){
  int *mat_1, *mat_2, *res; // CPU variables
  int *d_mat_1, *d_mat_2, *d_res; // GPU variables

  mat_1 = (int*) malloc(sizeof(int) * N * N); // Matrix 1
  mat_2 = (int*) malloc(sizeof(int) * N * N); // Matrix 2
  res = (int*) malloc(sizeof(int) * N * N); // Result Matrix

  // Allocate memory on GPU for each matrix
  hipMalloc((void**)&d_mat_1, sizeof(int) * N * N);
  hipMalloc((void**)&d_mat_2, sizeof(int) * N * N);
  hipMalloc((void**)&d_res, sizeof(int) * N * N);

  // Fill matrices
  fill_mat(mat_1);
  fill_mat(mat_2);

  // Copy CPU variables to GPU
  hipMemcpy(d_mat_1, mat_1, N * N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_mat_2, mat_2, N * N * sizeof(int), hipMemcpyHostToDevice);

  // Call function in GPU
  add<<< (N * N / TPB), TPB>>>(d_mat_1, d_mat_2, d_res, (N * N));

  // Copy result matrix from GPU to CPU
  hipMemcpy(res, d_res, N * N * sizeof(int), hipMemcpyDeviceToHost);

  //print_mat(res);
  printf("Done.\n");

  // Free CPU memory
  free(mat_1);
  free(mat_2);
  free(res);

  // Free GPU memory
  hipFree(d_mat_1);
  hipFree(d_mat_2);
  hipFree(d_res);

  return 0;
}
